
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void whoami(void) {
    int blockid = 
        blockIdx.x +
        blockIdx.y * gridDim.x +
        blockIdx.z * gridDim.y * gridDim.x;
                
    int block_offset =
        blockid *
        blockDim.x * blockDim.y * blockDim.z;

    int thread_offset =
        threadIdx.x +
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.y * blockDim.x;

    int id = block_offset + thread_offset;

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
    id,
    blockIdx.x, blockIdx.y, blockIdx.z, blockid,
    threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
}

int main(int argc, char** kwargs){
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4;
    
    int blocks_per_grid = b_x* b_y* b_z;
    int threads_per_block = t_x* t_y* t_z;

    printf("%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x,b_y,b_z);
    dim3 threadsPerBlock(t_x,t_y,t_z);

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}